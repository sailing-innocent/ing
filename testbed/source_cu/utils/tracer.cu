#include "hip/hip_runtime.h"
#include <testbed/cuda/utils/tracer.cuh>
#include <testbed/cuda/utils/ray.cuh>

ING_NAMESPACE_BEGIN

ING_CU_HOST_DEVICE color ray_color(const Ray& r)
{
    point center{0, 0, -1};
    float radius = 0.5f;
    vec4 oc = r.origin() - center;
    float a = dot(r.dir(), r.dir());
    float b = 2.0 * dot(oc, r.dir());
    float c = dot(oc, oc) - radius * radius;
    auto discriminant = b * b - 4 * a * c;
    if (discriminant > 0) {
        return color(1.0f, 0.0f, 0.0f);
    }

    // background is blended blue & white
    vec4 unit_direction = unit_vector(r.dir());
    auto t = 0.5 * (unit_direction.y() + 1.0);
    return (1.0 - t)*color(1.0f, 1.0f, 1.0f) + t * color(0.5, 0.7, 1.0);
}

__global__ void tracer_kernel(
        float* positions, 
        float time, 
        unsigned int width, 
        unsigned int height,
        World world
    )
{
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    // calculate uv coordinate
    float u = x / (float)width;
    float v = y / (float)height;

    float ou = 2 * u - 1;
    float ov = 2 * v - 1;

    float aspect_ratio = (float)width/(float)height;

    // camera
    float viewport_height = 2.0f;
    float viewport_width = aspect_ratio * viewport_height;
    float focal_length = 1.0f;

    point origin = point(0,0,0);
    auto horizontal = vec4(viewport_width, 0, 0, 0);
    auto vertical = vec4(0, viewport_height, 0, 0);
    point lower_left_corner = origin - horizontal/2 - vertical/2 - vec4(0, 0, focal_length, 0);

    // generate ray
    point ray_dir = lower_left_corner + u * horizontal + v * vertical - origin;
    Ray ray{origin, ray_dir};

    // if (world.hit(ray)) color = ray_color(world, ray)
    color c = ray_color(ray);

    // write vertex and color
    positions[8*(y*width+x)+0] = ou;
    positions[8*(y*width+x)+1] = ov;
    positions[8*(y*width+x)+2] = 0.0f;
    positions[8*(y*width+x)+3] = 1.0f;
    // color
    positions[8*(y*width+x)+4] = c[0];
    positions[8*(y*width+x)+5] = c[1];
    positions[8*(y*width+x)+6] = c[2];
    positions[8*(y*width+x)+7] = c[3];
}

ING_NAMESPACE_END