#include "hip/hip_runtime.h"
#include <testbed/cuda/utils/base.cuh>

__device__ float trans(float u) {
    return u * 2.0f - 1.0f;
}

__global__ void createVertices(float* positions, float time, unsigned int width, unsigned int height)
{
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

    // calculate uv coordinate
    float u = x / (float)width;
    float v = y / (float)height;

    u = trans(u);
    v = trans(v);

    // calculate simple sine wave pattern
    float freq = 18.0f;
    // float w = sinf(u * freq + time) * cosf(v * freq + time) * 0.5f;
    float w = sinf(freq * sqrtf( u * u + v * v) - time * 6.0f);
    // write position
    // positions[y*width + x] = make_float4(u, v, w, 1.0f);
    positions[8*(y*width+x)+0] = u;
    positions[8*(y*width+x)+1] = v;
    positions[8*(y*width+x)+2] = w;
    positions[8*(y*width+x)+3] = 1.0f;
    // generate color
    positions[8*(y*width+x)+4] = w + 0.5f;
    positions[8*(y*width+x)+5] = 0.3f;
    positions[8*(y*width+x)+6] = 0.8f;
    positions[8*(y*width+x)+7] = 1.0f;
}
